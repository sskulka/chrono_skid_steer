#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Eric Brandt, Asher Elmquist
// =============================================================================
//
// =============================================================================

#include <hip/hip_runtime.h>
#include "image_ops.cuh"
#include "chrono_sensor/optix/shaders/device_utils.h"
#include <iostream>

namespace chrono {
namespace sensor {

__global__ void image_gauss_kernel_vert(unsigned char* buf, int w, int h, int c, int f_width, float* dweights) {
    int index = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer

    // only run for each output pixel
    if (index < w * h * c) {
        // float f_std = (float)f / 2.f;
        // int f_width = (int)(2.f * 3.14f * f_std);

        int channel = index % c;
        int col = index / c % w;
        int row = index / c / w;

        float sum = 0;
        for (int i = -f_width; i <= f_width; i++) {
            int index_in = channel + col * c + abs(row + i) * w * c;
            if (row + i >= h)
                index_in = channel + col * c + (2 * h - (row + i + 1)) * w * c;

            // float weight = exp(-i * i / (2 * f_std * f_std)) / sqrtf(2.f * 3.14f * f_std * f_std);
            sum += dweights[i + f_width] * ((float)buf[index_in]);
            // sum += ((float)buf[index_in]);
        }
        buf[index] = (unsigned char)(sum);
    }
}

__global__ void image_gauss_kernel_horiz(unsigned char* buf, int w, int h, int c, int f_width, float* dweights) {
    int index = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer

    // only run for each output pixel
    if (index < w * h * c) {
        int channel = index % c;
        int col = index / c % w;
        int row = index / c / w;
        float sum = 0;
        for (int i = -f_width; i <= f_width; i++) {
            int index_in = channel + abs(col + i) * c + row * w * c;
            if (col + i >= w)
                index_in = channel + (2 * w - (col + i + 1)) * c + row * w * c;
            sum += dweights[i + f_width] * ((float)buf[index_in]);
        }
        buf[index] = (unsigned char)(sum);
    }
}

// merge pixels by the factor
__global__ void image_alias_kernel(unsigned char* bufIn,
                                   unsigned char* bufOut,
                                   int w_out,
                                   int h_out,
                                   int factor,
                                   int pix_size) {
    int out_index = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer

    int w_in = w_out * factor;
    int h_in = h_out * factor;
    //
    // // only run for each output pixel
    if (out_index < w_out * h_out * pix_size) {
        int idc_out = out_index % pix_size;
        int idx_out = (out_index / pix_size) % w_out;
        int idy_out = (out_index / pix_size) / w_out;

        float mean = 0.0;

        for (int i = -1; i < factor + 1; i++) {
            for (int j = -1; j < factor + 1; j++) {
                int idc_in = idc_out;
                int idx_in = idx_out * factor + j;
                int idy_in = idy_out * factor + i;

                // reflect when out of range

                if (idx_in < 0)
                    idx_in = -idx_in - 1;
                else if (idx_in >= w_in)
                    idx_in = 2 * w_in - (idx_in + 1);
                if (idy_in < 0)
                    idy_in = -idy_in - 1;
                else if (idy_in >= h_in)
                    idy_in = 2 * h_in - (idy_in + 1);

                int in_index = idy_in * w_in * pix_size + idx_in * pix_size + idc_in;
                mean += (float)bufIn[in_index];
            }
        }
        // bufOut[out_index] = (unsigned char)(mean / (factor * factor));
        bufOut[out_index] = (unsigned char)(mean / ((factor + 2) * (factor + 2)));
        if (idc_out == 3) {
            bufOut[out_index] = 255;
        }
        // bufOut[out_index] = (unsigned char)(25 * idc_out);
    }
}

// merge pixels by the factor
__global__ void image_alias_float_kernel(float* bufIn, float* bufOut, int w_out, int h_out, int factor, int pix_size) {
    int out_index = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer

    int w_in = w_out * factor;
    //
    // // only run for each output pixel
    if (out_index < w_out * h_out * pix_size) {
        int idc_out = out_index % pix_size;
        int idx_out = (out_index / pix_size) % w_out;
        int idy_out = (out_index / pix_size) / w_out;

        float mean = 0.0;

        for (int i = 0; i < factor; i++) {
            for (int j = 0; j < factor; j++) {
                int idc_in = idc_out;
                int idx_in = idx_out * factor + j;
                int idy_in = idy_out * factor + i;

                int in_index = idy_in * w_in * pix_size + idx_in * pix_size + idc_in;
                mean += bufIn[in_index];
            }
        }
        bufOut[out_index] = mean / (factor * factor);
    }
}
// merge pixels by the factor
__global__ void image_half4_to_uchar4_kernel(__half* bufIn, unsigned char* bufOut, int N) {
    int idx = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer
    if (idx < N) {
        bufOut[idx] = (unsigned char)(clamp(__half2float(bufIn[idx]), 0.f, 1.f) * 255.f);
    }
}

void cuda_image_gauss_blur_char(void* buf, int w, int h, int c, int factor, hipStream_t& stream) {
    const int nThreads = 512;
    int nBlocks = (w * h * c + nThreads - 1) / nThreads;

    float f_std = (float)factor / 4.f;
    int f_width = (int)(3.14f * f_std);

    int entries = 2 * f_width + 1;

    float* weights = new float[entries];

    for (int i = 0; i <= 2 * f_width; i++) {
        int offset = i - f_width;
        weights[i] = exp(-offset * offset / (2 * f_std * f_std)) / sqrtf(2.f * 3.14f * f_std * f_std);
    }
    float* dweights;
    hipMalloc(&dweights, entries * sizeof(float));
    hipMemcpy(dweights, weights, entries * sizeof(float), hipMemcpyHostToDevice);

    image_gauss_kernel_vert<<<nBlocks, nThreads, 0, stream>>>((unsigned char*)buf, w, h, c, f_width, dweights);
    image_gauss_kernel_horiz<<<nBlocks, nThreads, 0, stream>>>((unsigned char*)buf, w, h, c, f_width, dweights);
    hipFree(dweights);
    delete[] weights;
}

void cuda_image_alias(void* bufIn, void* bufOut, int w_out, int h_out, int factor, int pix_size, hipStream_t& stream) {
    const int nThreads = 512;
    int nBlocks = (w_out * h_out * pix_size + nThreads - 1) / nThreads;

    image_alias_kernel<<<nBlocks, nThreads, 0, stream>>>((unsigned char*)bufIn, (unsigned char*)bufOut, w_out, h_out,
                                                         factor, pix_size);
}

void cuda_image_alias_float(void* bufIn,
                            void* bufOut,
                            int w_out,
                            int h_out,
                            int factor,
                            int pix_size,
                            hipStream_t& stream) {
    const int nThreads = 512;
    int nBlocks = (w_out * h_out * pix_size + nThreads - 1) / nThreads;

    image_alias_float_kernel<<<nBlocks, nThreads, 0, stream>>>((float*)bufIn, (float*)bufOut, w_out, h_out, factor,
                                                               pix_size);
}

void cuda_image_half4_to_uchar4(void* bufIn, void* bufOut, int w, int h, hipStream_t& stream) {
    const int nThreads = 512;
    int nBlocks = (w * h * 4 + nThreads - 1) / nThreads;
    image_half4_to_uchar4_kernel<<<nBlocks, nThreads, 0, stream>>>((__half*)bufIn, (unsigned char*)bufOut, w * h * 4);
}

}  // namespace sensor
}  // namespace chrono
