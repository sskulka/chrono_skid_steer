#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// camera ray launch kernels
//
// =============================================================================

#include "chrono_sensor/optix/shaders/device_utils.h"

extern "C" __global__ void __raygen__camera_pinhole() {
    const RaygenParameters* raygen = (RaygenParameters*)optixGetSbtDataPointer();
    const CameraParameters& camera = raygen->specific.camera;

    const uint3 idx = optixGetLaunchIndex();
    const uint3 screen = optixGetLaunchDimensions();
    const unsigned int image_index = screen.x * idx.y + idx.x;

    float2 d =
        (make_float2(idx.x, idx.y) + make_float2(0.5, 0.5)) / make_float2(screen.x, screen.y) * 2.f - make_float2(1.f);
    d.y *= (float)(screen.y) / (float)(screen.x);  // correct for the aspect ratio

    const float t_frac = 0;  // 0-1 between start and end time of the camera (chosen here)
    const float t_traverse = raygen->t0 + t_frac * (raygen->t1 - raygen->t0);  // simulation time when ray is sent
    float3 ray_origin = lerp(raygen->pos0, raygen->pos1, t_frac);
    float4 ray_quat = nlerp(raygen->rot0, raygen->rot1, t_frac);
    const float h_factor = camera.hFOV / HIP_PI_F * 2.0;
    float3 forward;
    float3 left;
    float3 up;

    basis_from_quaternion(ray_quat, forward, left, up);
    float3 ray_direction = normalize(forward - d.x * left * h_factor + d.y * up * h_factor);

    PerRayData_camera prd = default_camera_prd();
    prd.use_gi = camera.use_gi;
    if (camera.use_gi) {
        prd.rng = camera.rng_buffer[image_index];
    }
    unsigned int opt1;
    unsigned int opt2;
    pointer_as_ints(&prd, opt1, opt2);
    unsigned int raytype = (unsigned int)CAMERA_RAY_TYPE;
    optixTrace(params.root, ray_origin, ray_direction, params.scene_epsilon, 1e16f, t_traverse, OptixVisibilityMask(1),
               OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);

    // Gamma correct the output color into sRGB color space
    float recip_gamma = 1 / camera.gamma;
    camera.frame_buffer[image_index] =
        make_half4(pow(prd.color.x, recip_gamma), pow(prd.color.y, recip_gamma), pow(prd.color.z, recip_gamma), 1.f);
    if (camera.use_gi) {
        camera.albedo_buffer[image_index] = make_half4(prd.albedo.x, prd.albedo.y, prd.albedo.z, 0.f);
        float screen_n_x = -Dot(left, prd.normal);     // screen space (x right)
        float screen_n_y = Dot(up, prd.normal);        // screen space (y up)
        float screen_n_z = -Dot(forward, prd.normal);  // screen space (-z forward)
        camera.normal_buffer[image_index] = make_half4(screen_n_x, screen_n_y, screen_n_z, 0.f);
    }
}

/// Camera ray generation program using an FOV lens model
extern "C" __global__ void __raygen__camera_fov_lens() {
    const RaygenParameters* raygen = (RaygenParameters*)optixGetSbtDataPointer();
    const CameraParameters& camera = raygen->specific.camera;

    const uint3 idx = optixGetLaunchIndex();
    const uint3 screen = optixGetLaunchDimensions();
    const unsigned int image_index = screen.x * idx.y + idx.x;

    float2 d =
        (make_float2(idx.x, idx.y) + make_float2(0.5, 0.5)) / make_float2(screen.x, screen.y) * 2.f - make_float2(1.f);
    d.y *= (float)(screen.y) / (float)(screen.x);  // correct for the aspect ratio

    if (abs(d.x) > 1e-5 || abs(d.y) > 1e-5) {
        float r1 = sqrtf(d.x * d.x + d.y * d.y);
        float r2 = tanf(r1 * tanf(camera.hFOV / 2.0)) / tanf(camera.hFOV / 2.0);
        float scaled_extent = tanf(tanf(camera.hFOV / 2.0)) / tanf(camera.hFOV / 2.0);
        d.x = d.x * (r2 / r1) / scaled_extent;
        d.y = d.y * (r2 / r1) / scaled_extent;
    }

    const float t_frac = 0;  // 0-1 between start and end time of the camera (chosen here)
    const float t_traverse = raygen->t0 + t_frac * (raygen->t1 - raygen->t0);  // simulation time when ray is sent
    float3 ray_origin = lerp(raygen->pos0, raygen->pos1, t_frac);
    float4 ray_quat = nlerp(raygen->rot0, raygen->rot1, t_frac);
    const float h_factor = camera.hFOV / HIP_PI_F * 2.0;
    float3 forward;
    float3 left;
    float3 up;

    basis_from_quaternion(ray_quat, forward, left, up);
    float3 ray_direction = normalize(forward - d.x * left * h_factor + d.y * up * h_factor);

    PerRayData_camera prd = default_camera_prd();
    prd.use_gi = camera.use_gi;
    if (camera.use_gi) {
        prd.rng = camera.rng_buffer[image_index];
    }
    unsigned int opt1;
    unsigned int opt2;
    pointer_as_ints(&prd, opt1, opt2);
    unsigned int raytype = (unsigned int)CAMERA_RAY_TYPE;
    optixTrace(params.root, ray_origin, ray_direction, params.scene_epsilon, 1e16f, t_traverse, OptixVisibilityMask(1),
               OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);

    // Gamma correct the output color into sRGB color space
    float gamma = camera.gamma;
    camera.frame_buffer[image_index] =
        make_half4(pow(prd.color.x, 1.0f / gamma), pow(prd.color.y, 1.0f / gamma), pow(prd.color.z, 1.0f / gamma), 1.f);
    if (camera.use_gi) {
        camera.albedo_buffer[image_index] = make_half4(prd.albedo.x, prd.albedo.y, prd.albedo.z, 0.f);
        float screen_n_x = -Dot(left, prd.normal);     // screen space (x right)
        float screen_n_y = Dot(up, prd.normal);        // screen space (y up)
        float screen_n_z = -Dot(forward, prd.normal);  // screen space (-z forward)
        camera.normal_buffer[image_index] = make_half4(screen_n_x, screen_n_y, screen_n_z, 0.f);
    }
}

extern "C" __global__ void __raygen__segmentation_pinhole() {
    const RaygenParameters* raygen = (RaygenParameters*)optixGetSbtDataPointer();
    const SemanticCameraParameters& camera = raygen->specific.segmentation;

    const uint3 idx = optixGetLaunchIndex();
    const uint3 screen = optixGetLaunchDimensions();
    const unsigned int image_index = screen.x * idx.y + idx.x;

    float2 d =
        (make_float2(idx.x, idx.y) + make_float2(0.5, 0.5)) / make_float2(screen.x, screen.y) * 2.f - make_float2(1.f);
    d.y *= (float)(screen.y) / (float)(screen.x);  // correct for the aspect ratio

    const float t_frac = 0;  // 0-1 between start and end time of the camera (chosen here)
    const float t_traverse = raygen->t0 + t_frac * (raygen->t1 - raygen->t0);  // simulation time when ray is sent
    float3 ray_origin = lerp(raygen->pos0, raygen->pos1, t_frac);
    float4 ray_quat = nlerp(raygen->rot0, raygen->rot1, t_frac);
    const float h_factor = camera.hFOV / HIP_PI_F * 2.0;
    float3 forward;
    float3 left;
    float3 up;

    basis_from_quaternion(ray_quat, forward, left, up);
    float3 ray_direction = normalize(forward - d.x * left * h_factor + d.y * up * h_factor);

    PerRayData_semantic prd = default_semantic_prd();
    unsigned int opt1;
    unsigned int opt2;
    pointer_as_ints(&prd, opt1, opt2);
    unsigned int raytype = (unsigned int)SEGMENTATION_RAY_TYPE;
    optixTrace(params.root, ray_origin, ray_direction, params.scene_epsilon, 1e16f, t_traverse, OptixVisibilityMask(1),
               OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);

    camera.frame_buffer[image_index].x = prd.class_id;
    camera.frame_buffer[image_index].y = prd.instance_id;
}

/// Camera ray generation program using an FOV lens model
extern "C" __global__ void __raygen__segmentation_fov_lens() {
    const RaygenParameters* raygen = (RaygenParameters*)optixGetSbtDataPointer();
    const SemanticCameraParameters& camera = raygen->specific.segmentation;

    const uint3 idx = optixGetLaunchIndex();
    const uint3 screen = optixGetLaunchDimensions();
    const unsigned int image_index = screen.x * idx.y + idx.x;

    float2 d =
        (make_float2(idx.x, idx.y) + make_float2(0.5, 0.5)) / make_float2(screen.x, screen.y) * 2.f - make_float2(1.f);
    d.y *= (float)(screen.y) / (float)(screen.x);  // correct for the aspect ratio

    if (abs(d.x) > 1e-5 || abs(d.y) > 1e-5) {
        float r1 = sqrtf(d.x * d.x + d.y * d.y);
        float r2 = tanf(r1 * tanf(camera.hFOV / 2.0)) / tanf(camera.hFOV / 2.0);
        float scaled_extent = tanf(tanf(camera.hFOV / 2.0)) / tanf(camera.hFOV / 2.0);
        d.x = d.x * (r2 / r1) / scaled_extent;
        d.y = d.y * (r2 / r1) / scaled_extent;
    }

    const float t_frac = 0;  // 0-1 between start and end time of the camera (chosen here)
    const float t_traverse = raygen->t0 + t_frac * (raygen->t1 - raygen->t0);  // simulation time when ray is sent
    float3 ray_origin = lerp(raygen->pos0, raygen->pos1, t_frac);
    float4 ray_quat = nlerp(raygen->rot0, raygen->rot1, t_frac);
    const float h_factor = camera.hFOV / HIP_PI_F * 2.0;
    float3 forward;
    float3 left;
    float3 up;

    basis_from_quaternion(ray_quat, forward, left, up);
    float3 ray_direction = normalize(forward - d.x * left * h_factor + d.y * up * h_factor);

    PerRayData_semantic prd = default_semantic_prd();
    unsigned int opt1;
    unsigned int opt2;
    pointer_as_ints(&prd, opt1, opt2);
    unsigned int raytype = (unsigned int)SEGMENTATION_RAY_TYPE;
    optixTrace(params.root, ray_origin, ray_direction, params.scene_epsilon, 1e16f, t_traverse, OptixVisibilityMask(1),
               OPTIX_RAY_FLAG_NONE, 0, 1, 0, opt1, opt2, raytype);
    camera.frame_buffer[image_index].x = prd.class_id;
    camera.frame_buffer[image_index].y = prd.instance_id;
}
